
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

__global__ void staticReverse(int *d, int n)
{
	__shared__ int s[64]; //static shared memory allocation
	int t = threadIdx.x; 
	int tr = n - threadIdx.x - 1;
	if(t < n)
	{
		s[t] = d[t]; 
		__syncthreads(); //None shall pass
		d[t] = s[tr];
	}
}

__global__ void dynamicReverse(int *d, int n)
{
	extern __shared__ int s[]; 
	int t = threadIdx.x;
	int tr = n-t-1; 
	if(t < n)
	{
		s[t] = d[t];
		__syncthreads();
		d[t] = s[tr];
	}
}

int main()
{
	const int n = 64; 
	int *a, *d_a; 
	a = (int*)malloc(n*sizeof(int));
	hipMalloc(&d_a, n*sizeof(int)); 

	for(int i =0; i < n; i++)
		a[i] = i; 
	
	hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice); // Transfer to device

	dynamicReverse<<<1, n, n*sizeof(int)>>>(d_a, n); //grid ,block ,shared
	
	hipMemcpy(a,d_a, n*sizeof(int), hipMemcpyDeviceToHost); //bring it back
	
	std::cout<<a[0]<<std::endl;
	free(a);
	hipFree(d_a);
	return 0;
}
