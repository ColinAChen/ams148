
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <stdlib.h>
#include <stdio.h>

#define M 128


__global__ void ftcs(float *f, const float dx, const float k, const float dt)
{
	extern __shared__ float temp[];
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int id = threadIdx.x;
	if(tid >= M)
	{
		return;
	}

	temp[id] = f[tid];// load data into temp; 
	__syncthreads();
 
	if(id > 0 && id < blockDim.x-1)
	{
		float temp2 =  temp[id] + k*dt/(dx*dx)*(temp[id+1] - 2*temp[id] + temp[id-1]); 
		f[tid] = temp2;	
	}

}

__global__ void bc(float *f)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x; 
	if(tid == 0) //use only one thread for 1D BC
	{
		f[0] = f[1];
		f[M-1] = f[M-2]; 
	}
}

void io_fun(std::string file, float *x, float *f)
{
	std::ofstream myfile_tsN; 
	myfile_tsN.open(file); 
	for(int i = 0; i < M; i++)
	{
		myfile_tsN << x[i] << '\t';
		myfile_tsN << f[i] << std::endl;
	}	

	myfile_tsN.close(); 
}


int main()
{
	float k = 1.0f; 
	float dx = 2.0f/float(M); 
	float dt = 0.5f*(dx*dx)/k; 
	float x[M];
	float tmax = 2.0f; 
	float t = 0.0f, tio = 0.5f; 
	
	//Allocate Memory 
	size_t sz = M*sizeof(float); 
	float *f; 
	f = (float*)malloc(sz); 
	float *d_f; 
	hipMalloc(&d_f, sz);
	
	//Kernel parameters
	dim3 dimBlock(16,1,1); 
	dim3 dimGrid(M/dimBlock.x, 1,1); 
	
	//Apply Initial Condition You could also create a kernel for this
	for(int i=0; i < M; i++)
	{
		x[i] = -1.0f + i*dx; 
		f[i] = exp(-0.5f*pow(x[i],2));
	}

	//Transfer to device
	hipMemcpy(d_f, f, sz, hipMemcpyHostToDevice); 

	/* IO Operations for IC */
	std::string f1 = "IC.dat"; 
	io_fun(f1, x, f);
	
	/*Perform Integration */ 

	std::string f2;
	int kk = 0;
	while(t<tmax)
	{
	//Call the stencil routine
		ftcs<<<dimGrid, dimBlock, dimBlock.x*sizeof(float)>>>(d_f, dx, k, dt); 
		hipDeviceSynchronize(); 
	//Call BC
//		bc<<<dimGrid, dimBlock>>>(d_f); 
//		cudaDeviceSynchronize();
		if(fmod(t, tio) == 0.0f)
		{
		//IO function
			f2 = "sol" + std::to_string(kk) + ".dat"; 
			hipMemcpy(f,d_f, sz, hipMemcpyDeviceToHost);
			io_fun(f2, x, f); 
		}

		t+=dt;
		kk++;
	}

	if(fmod(tmax,tio) != 0.0f)
	{//IO Function 
		f2 = "final_sol.dat"; 
		hipMemcpy(f,d_f, sz, hipMemcpyDeviceToHost);
		io_fun(f2, x, f); 
	}

	//deallocate memory 
	free(f); 
	hipFree(d_f); 
}

