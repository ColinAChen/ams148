
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <stdlib.h>
#include <stdio.h>

#define M 1024 


__global__ void ftcs(float *f, const float dx, const float k, const float dt)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if(tid > 0 && tid < M-1)
	{
		float temp2 =  f[tid] + k*dt/(dx*dx)*(f[tid+1] - 2*f[tid] + f[tid-1]); 
		f[tid] = temp2;	
	}

}

__global__ void bc(float *f)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x; 
	if(tid == 0) //use only one thread for 1D BC
	{
		f[0] = f[1];
		f[M-1] = f[M-2]; 
	}
}

void io_fun(std::string file, float *x, float *f)
{
	std::ofstream myfile_tsN; 
	myfile_tsN.open(file); 
	for(int i = 0; i < M; i++)
	{
		myfile_tsN << x[i] << '\t';
		myfile_tsN << f[i] << std::endl;
	}	

	myfile_tsN.close(); 
}


int main()
{
	float k = 1.0f; 
	float dx = 2.0f/float(M); 
	float dt = 0.5f*(dx*dx)/k; 
	float x[M];
	float tmax = 0.5f; 
	float t = 0.0f, tio = 0.125f; 
	
	//Allocate Memory 
	size_t sz = M*sizeof(float); 
	float *f; 
	f = (float*)malloc(sz); 
	float *d_f; 
	hipMalloc(&d_f, sz);
	
	//Kernel parameters
	dim3 dimBlock(16,1,1); 
	dim3 dimGrid(M/dimBlock.x, 1,1); 
	
	//Apply Initial Condition You could also create a kernel for this
	for(int i=0; i < M; i++)
	{
		x[i] = -1.0f + i*dx; 
		f[i] = exp(-0.5f*pow(x[i],2));
	//	f[i] = 1.0;
	}

	//Transfer to device
	hipMemcpy(d_f, f, sz, hipMemcpyHostToDevice); 

	/* IO Operations for IC */
	std::string f1 = "IC.dat"; 
	io_fun(f1, x, f);
	
	/*Perform Integration */ 

	std::string f2;
	int kk = 0;
	while(t<tmax)
	{
	//Call the stencil routine
		ftcs<<<dimGrid, dimBlock>>>(d_f, dx, k, dt); 
		hipDeviceSynchronize(); 
	//Call BC
		bc<<<dimGrid, dimBlock>>>(d_f); 
		hipDeviceSynchronize();
		if(fmod(t, tio) == 0.0f)
		{
		//IO function
			f2 = "sol" + std::to_string(kk) + ".dat"; 
			hipMemcpy(f,d_f, sz, hipMemcpyDeviceToHost);
			io_fun(f2, x, f); 
		}

		t+=dt;
		kk++;
	}

	if(fmod(tmax,tio) != 0.0f)
	{//IO Function 
		f2 = "final_sol.dat"; 
		hipMemcpy(f,d_f, sz, hipMemcpyDeviceToHost);
		io_fun(f2, x, f); 
	}

	//deallocate memory 
	free(f); 
	hipFree(d_f); 
}

